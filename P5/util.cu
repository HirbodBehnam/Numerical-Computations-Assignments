
#include <hip/hip_runtime.h>
#define REDUCTION_OUTPUT_SIZE 1

__global__ void reduce_sum_device(const float *g_idata, float *g_odata) {
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}

/***
 * Finds the next power of two of a number.
 * https://stackoverflow.com/a/2681094
 * @param x The number to find its next power of two.
 * @return The next power of two.
 */
unsigned int prev_power_of_two(unsigned int x) {
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    return x - (x >> 1);
}

float reduce_sum(float *device_mem, unsigned int size) {
    // Find the previous power of two
    unsigned int power_of_two_size = prev_power_of_two(size);
    float *output_reduction;
    hipMalloc(&output_reduction, sizeof(float) * REDUCTION_OUTPUT_SIZE);
    // Reduce
    reduce_sum_device<<<REDUCTION_OUTPUT_SIZE, power_of_two_size / REDUCTION_OUTPUT_SIZE, power_of_two_size / REDUCTION_OUTPUT_SIZE>>>(device_mem,
                                                                                                      output_reduction);
    // Move back result to CPU
    float reduction_result[REDUCTION_OUTPUT_SIZE];
    hipMemcpy(reduction_result, output_reduction, sizeof(float) * REDUCTION_OUTPUT_SIZE, hipMemcpyDeviceToHost);
    hipFree(output_reduction);
    // Final reduction on CPU
    float result = 0;
    for (float i: reduction_result)
        result += i;
    // Also the rest of array
    unsigned int rest_size = size - power_of_two_size;
    auto *rest_of_array = new float[rest_size];
    hipMemcpy(rest_of_array, device_mem + power_of_two_size, sizeof(float) * rest_size, hipMemcpyDeviceToHost);
    for (unsigned int i = 0; i < rest_size; i++) {
        result += rest_of_array[i];
    }
    delete[] rest_of_array;
    hipFree(output_reduction);
    return result;
}
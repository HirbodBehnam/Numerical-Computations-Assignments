
#include <hip/hip_runtime.h>
__global__ void five_point(const float *y_terms, float *result, float h) {
    unsigned int size = blockDim.x;
    unsigned int i = threadIdx.x;
    if (i < 2 || i >= size - 2) {
        return;
    }
    result[i] = (y_terms[i - 2] - 8 * y_terms[i - 1] + 8 * y_terms[i + 1] - y_terms[i + 2]) / (12 * h);
}

void differentiate(float *xs, float *fxs, int length, float *out) {
    // Move everything to cuda memory
    float *fxs_gpu, *result;
    hipMalloc(&fxs_gpu, sizeof(float) * length);
    hipMemcpy(fxs_gpu, fxs, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMalloc(&result, sizeof(float) * length);
    // Calculate the points
    five_point<<<1, length>>>(fxs_gpu, result, xs[1] - xs[0]);
    hipMemcpy(out, result, sizeof(float) * length, hipMemcpyDeviceToHost);
    // Clean up
    hipFree(fxs_gpu);
    hipFree(result);
}
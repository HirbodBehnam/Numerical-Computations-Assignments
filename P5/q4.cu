#include "hip/hip_runtime.h"
#include "util.cuh"

__global__ void simpson_13_method(const float *y_terms, float *result) {
    unsigned int size = blockDim.x;
    unsigned int i = threadIdx.x;
    if (threadIdx.x == 0 || threadIdx.x == size - 1) {
        result[i] = y_terms[i];
    } else if (i % 2 == 1) {
        result[i] = 4 * y_terms[i];
    } else {
        result[i] = 2 * y_terms[i];
    }
}

float integrate(float start, float end, float *xs, float *fxs, int length) {
    // Move everything to cuda memory
    float *fxs_gpu, *simpson_array;
    hipMalloc(&fxs_gpu, sizeof(float) * length);
    hipMemcpy(fxs_gpu, fxs, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMalloc(&simpson_array, sizeof(float) * length);
    float h = xs[1] - xs[0];
    // Do the calculations
    simpson_13_method<<<1, length>>>(fxs_gpu, simpson_array);
    float result = reduce_sum(simpson_array, length);
    // Cleanup
    hipFree(fxs_gpu);
    hipFree(simpson_array);
    return h * result / 3;
}

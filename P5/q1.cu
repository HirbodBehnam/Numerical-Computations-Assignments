#include "hip/hip_runtime.h"
#include <iostream>
#include "util.cuh"

__global__ void calculate_terms(float *coefficients, float x) {
    unsigned int tid = threadIdx.x;
    coefficients[tid] *= powf(x, static_cast<float>(tid));
}

float calculatePolynomial(const float *coefficients, int length, float x) {
    // Move everything to cuda memory
    float *coefficients_gpu;
    hipMalloc(&coefficients_gpu, sizeof(float) * length);
    hipMemcpy(coefficients_gpu, coefficients, sizeof(float) * length, hipMemcpyHostToDevice);
    // Calculate the terms
    calculate_terms<<<1, length>>>(coefficients_gpu, x);
    float result = reduce_sum(coefficients_gpu, length);
    hipFree(coefficients_gpu);
    return result;
}
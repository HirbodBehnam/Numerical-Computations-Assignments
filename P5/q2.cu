#include "hip/hip_runtime.h"
#include "util.cuh"
#include <stdio.h>

__global__ void calculate_l_terms(const float *x_terms, float *l_terms, float x) {
    unsigned int j = threadIdx.x;
    unsigned int k = blockDim.x;
    float result = 1;
    for (unsigned int m = 0; m < k; m++) {
        if (m == j)
            continue;
        result *= (x - x_terms[m]) / (x_terms[j] - x_terms[m]);
    }
    l_terms[j] = result;
}

__global__ void calculate_ly_terms(const float *y_terms, float *l_terms) {
    unsigned int i = threadIdx.x;
    l_terms[i] *= y_terms[i];
}

__global__ void divided_differences(const float *x_terms, const float *y_terms, float *result, float x) {
    extern __shared__ float prev_data[];
    extern __shared__ float current_data[];
    unsigned int i = threadIdx.x;
    prev_data[i] = y_terms[i];
    __syncthreads();

    for (unsigned int j = 0; j < i; j++) {
        current_data[i] = (prev_data[i] - prev_data[i - 1]) / (x_terms[i]- x_terms[i - j - 1]);
        __syncthreads();
        prev_data[i] = current_data[i];
        __syncthreads();
    }

    result[i] = prev_data[i];
    for (unsigned int j = 0; j < i; j++) {
        result[i] *= (x - x_terms[j]);
    }
}

float lagrangeInterpolate(float *xs, float *fxs, int length, float x) {
    // Move everything to cuda memory
    float *xs_gpu, *fxs_gpu, *l_terms;
    hipMalloc(&xs_gpu, sizeof(float) * length);
    hipMemcpy(xs_gpu, xs, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMalloc(&fxs_gpu, sizeof(float) * length);
    hipMemcpy(fxs_gpu, fxs, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMalloc(&l_terms, sizeof(float) * length);
    // Calculate the terms
    calculate_l_terms<<<1, length>>>(xs_gpu, l_terms, x);
    calculate_ly_terms<<<1, length>>>(fxs_gpu, l_terms);
    float result = reduce_sum(l_terms, length);
    // Cleanup
    hipFree(xs_gpu);
    hipFree(fxs_gpu);
    hipFree(l_terms);
    return result;
}

float newtonInterpolate(float *xs, float *fxs, int length, float x) {
    // Move everything to cuda memory
    float *xs_gpu, *fxs_gpu, *divided_differences_terms;
    hipMalloc(&xs_gpu, sizeof(float) * length);
    hipMemcpy(xs_gpu, xs, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMalloc(&fxs_gpu, sizeof(float) * length);
    hipMemcpy(fxs_gpu, fxs, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMalloc(&divided_differences_terms, sizeof(float) * length);
    // Calculate terms
    divided_differences<<<1, length, length>>>(xs_gpu, fxs_gpu, divided_differences_terms, x);
    float result = reduce_sum(divided_differences_terms, length);
    // Cleanup
    hipFree(xs_gpu);
    hipFree(fxs_gpu);
    hipFree(divided_differences_terms);
    return result;
}